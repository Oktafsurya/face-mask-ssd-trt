#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include "opencv2/imgproc/imgproc.hpp"
#include <stdbool.h>
#include <math.h>

using namespace cv;
using namespace std;

#define BLOCKSIZE 32 //The number of threads per block should be not greater than 1024

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

typedef struct
{
    float x,y,w,h,s;

}box;

__device__
float IOUcalc(box b1, box b2)
{
    float ai = (float)(b1.w)*(b1.h);
    float aj = (float)(b2.w)*(b2.h);
    float x_inter, x2_inter, y_inter, y2_inter;

    x_inter = max(b1.x,b2.x);
    y_inter = max(b1.y,b2.y);

    x2_inter = min((b1.x + b1.w),(b2.x + b2.w));
    y2_inter = min((b1.y + b1.h),(b2.y + b2.h));

    float w = (float)max((float)0, x2_inter - x_inter);  
    float h = (float)max((float)0, y2_inter - y_inter);  

    float inter = ((w*h)/(ai + aj - w*h));
    return inter;
}


__global__
void NMS_GPU(box *d_b, bool *d_res)
{
    int abs_y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int abs_x = (blockIdx.x * blockDim.x) +threadIdx.x;


    float theta = 0.6;


    if(d_b[abs_x].s < d_b[abs_y].s)
    {
        if(IOUcalc(d_b[abs_y],d_b[abs_x])>theta)
        {
            d_res[abs_x] = false; 
        }
    }
  
}

int main()
{
    int count = 6000;

    bool *h_res =(bool *)malloc(sizeof(bool)*count);

    for(int i=0; i<count; i++)
    {
        h_res[i] = true;
    }

    box b[count];
    
    std::ifstream in;
    std::string line;
    
    in.open("../boxes.txt"); //y1, x1, y2, x2
    if (in.is_open()) 
    {
        int i = 0;
        while(getline(in, line))
        {
            istringstream iss(line);
            iss >> b[i].y;
            iss >> b[i].x;
            iss >> b[i].h; //y2
            iss >> b[i].w; //x2
            b[i].h-=b[i].y; //y2 -> h
            b[i].w-=b[i].x; //x2 -> w
            i+=1;
            if(i==count) break;
        }
    }
    in.close();
    
    in.open("../scores.txt");
    if (in.is_open()) 
    {
        int i = 0;
        while(in >> b[i].s)
        {
            i+=1;
            if(i==count) break;
        }
    }
    in.close();
    
    box *d_b;
    bool *d_res;

    gpuErrchk(hipMalloc((void**)&d_res, count*sizeof(bool)));
    gpuErrchk(hipMemcpy(d_res, h_res,sizeof(bool)*count, hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc((void**)&d_b,sizeof(box)*count));
    gpuErrchk(hipMemcpy(d_b, b,sizeof(box)*count, hipMemcpyHostToDevice));
    
    //Setting 1: can only work when count <= 1024
    //NMS_GPU<<<dim3(1,count,1),count>>>(d_b,d_res);
    
    //Setting 2: work when count > 1024
    //NMS_GPU<<<dim3(count,count,1), 1>>>(d_b,d_res);
    
    //Setting 3: work when count > 1024, faster than Setting 2
    dim3 gridSize(int(ceil(float(count)/BLOCKSIZE)), int(ceil(float(count)/BLOCKSIZE)),1);
    dim3 blockSize(BLOCKSIZE, BLOCKSIZE, 1);
    NMS_GPU<<<gridSize, blockSize>>>(d_b,d_res);
    
    hipDeviceSynchronize();

    gpuErrchk(hipMemcpy(h_res, d_res, sizeof(bool)*count, hipMemcpyDeviceToHost));

    printf("Suppressed box id:\n");
    for(int i =0; i<count ; i++)
    {

        if(*(h_res+i) != true)
        {
            printf("%d ",i);
        }
        
    }

    return 0;
}